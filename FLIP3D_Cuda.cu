#include "hip/hip_runtime.h"
#include "FLIP3D_Cuda.cuh"
#define VEL 0
#define PRESS 0
#define LEVEL 0
#define DENSITY 0
#define DIV 0
#define CONTENT 0

FLIP3D_Cuda::FLIP3D_Cuda()
{

}

FLIP3D_Cuda:: ~FLIP3D_Cuda()
{
	FreeDeviceMem();
}

void FLIP3D_Cuda::Init(void)
{
	_wallThick = 1.0 / _gridRes;
	_cellPhysicalSize = 1.0 / _gridRes;

	_grid = new FLIPGRID(_gridRes, _cellPhysicalSize);

	ParticleInit();
	_numParticles = h_CurPos.size();
	cout << _numParticles << endl;

	//For grid visualize
	h_gridPos.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1)); 
	h_gridVel.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1)); 
	h_gridPress.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1)); 
	h_gridDens.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1));
	h_gridLevelSet.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1));
	h_gridDiv.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1));
	h_gridContent.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1)); 

	InitDeviceMem();
	CopyToDevice();

	ComputeWallParticleNormal_kernel();
	//hipDeviceSynchronize();
}

void FLIP3D_Cuda::ParticleInit()
{
	PlaceObjects();

	// Place Fluid Particles
	double w = _dens * _wallThick;
	for (int i = 0; i < _gridRes / _dens; i++) {
		for (int j = 0; j < _gridRes / _dens; j++) {
			for (int k = 0; k < _gridRes / _dens; k++) {
				double x = i * w + w / 2.0;
				double y = j * w + w / 2.0;
				double z = k * w + w / 2.0;
				
				if (x > _wallThick && x < 1.0 - _wallThick &&
					y > _wallThick && y < 1.0 - _wallThick &&
					z > _wallThick && z < 1.0 - _wallThick) {
					PushParticle(x, y, z, FLUID);
				}
			}
		}
	}

	// Place Wall Particles
	w = 1.0 / _gridRes;
	for (int i = 0; i < _gridRes; i++) {
		for (int j = 0; j < _gridRes; j++) {
			for (int k = 0; k < _gridRes; k++) {
				double x = i * w + w / 2.0;
				double y = j * w + w / 2.0;
				double z = k * w + w / 2.0;
				PushParticle(x, y, z, WALL);
			}
		}
	}

	for (int iter = 0; iter < h_CurPos.size(); iter++)
	{
		if (h_Type[iter] == WALL) {
			iter++;
			continue;
		}
		int i = fmin(_gridRes - 1, fmax(0, h_CurPos[iter].x * _gridRes));
		int j = fmin(_gridRes - 1, fmax(0, h_CurPos[iter].y * _gridRes));
		int k = fmin(_gridRes - 1, fmax(0, h_CurPos[iter].z * _gridRes));
	}
}

void FLIP3D_Cuda::PlaceObjects()
{
	PlaceWalls();

	WaterDropTest();
	//DamBreakTest();
}

void FLIP3D_Cuda::PlaceWalls()
{
	Object obj;

	// Left Wall
	obj.type = WALL;
	obj.shape = BOX;
	obj.material = GLASS;
	obj.visible = 0;
	obj.p[0].x = 0.0;			obj.p[1].x = _wallThick; //Box min, max ��
	obj.p[0].y = 0.0;			obj.p[1].y = 1.0;
	obj.p[0].z = 0.0;			obj.p[1].z = 1.0;
	objects.push_back(obj);

	// Right Wall
	obj.type = WALL;
	obj.shape = BOX;
	obj.material = GLASS;
	obj.visible = 0;
	obj.p[0].x = 1.0 - _wallThick;	obj.p[1].x = 1.0;
	obj.p[0].y = 0.0;				obj.p[1].y = 1.0;
	obj.p[0].z = 0.0;				obj.p[1].z = 1.0;
	objects.push_back(obj);

	// Floor Wall
	obj.type = WALL;
	obj.shape = BOX;
	obj.material = GLASS;
	obj.visible = 0;
	obj.p[0].x = 0.0;	obj.p[1].x = 1.0;
	obj.p[0].y = 0.0;	obj.p[1].y = _wallThick;
	obj.p[0].z = 0.0;	obj.p[1].z = 1.0;
	objects.push_back(obj);

	// Ceiling Wall
	obj.type = WALL;
	obj.shape = BOX;
	obj.material = GLASS;
	obj.visible = 0;
	obj.p[0].x = 0.0;				obj.p[1].x = 1.0;
	obj.p[0].y = 1.0 - _wallThick;	obj.p[1].y = 1.0;
	obj.p[0].z = 0.0;				obj.p[1].z = 1.0;
	objects.push_back(obj);

	// Front Wall
	obj.type = WALL;
	obj.shape = BOX;
	obj.material = GLASS;
	obj.visible = 0;
	obj.p[0].x = 0.0;	obj.p[1].x = 1.0;
	obj.p[0].y = 0.0;	obj.p[1].y = 1.0;
	obj.p[0].z = 0.0;	obj.p[1].z = _wallThick;
	objects.push_back(obj);

	// Back Wall
	obj.type = WALL;
	obj.shape = BOX;
	obj.material = GLASS;
	obj.visible = 0;
	obj.p[0].x = 0.0;				obj.p[1].x = 1.0;
	obj.p[0].y = 0.0;				obj.p[1].y = 1.0;
	obj.p[0].z = 1.0 - _wallThick;	obj.p[1].z = 1.0;
	objects.push_back(obj);
}

void FLIP3D_Cuda::WaterDropTest()
{
	Object obj;

	obj.type = FLUID;
	obj.shape = BOX;
	obj.p[0].x = _wallThick;	obj.p[1].x = 1.0 - _wallThick;
	obj.p[0].y = _wallThick;	obj.p[1].y = 0.2;
	obj.p[0].z = _wallThick;	obj.p[1].z = 1.0 - _wallThick;
	objects.push_back(obj);

	//obj.type = FLUID;
	//obj.shape = SPHERE;
	//obj.c.x = 0.5;
	//obj.c.y = 0.6;
	//obj.c.z = 0.5;
	//obj.r = 0.12;
	//objects.push_back(obj);
}

void FLIP3D_Cuda::DamBreakTest()
{
	Object obj;

	obj.type = FLUID;
	obj.shape = BOX;
	obj.visible = true;
	obj.p[0].x = 0.2;	obj.p[1].x = 0.4;
	obj.p[0].y = _wallThick;	obj.p[1].y = 0.4;
	obj.p[0].z = 0.2;	obj.p[1].z = 0.8;

	objects.push_back(obj);

	obj.type = FLUID;
	obj.shape = BOX;
	obj.visible = true;
	obj.p[0].x = _wallThick;	obj.p[1].x = 1.0 - _wallThick;
	obj.p[0].y = _wallThick;	obj.p[1].y = 0.06;
	obj.p[0].z = _wallThick;	obj.p[1].z = 1.0 - _wallThick;

	objects.push_back(obj);
}

void FLIP3D_Cuda::PushParticle(REAL x, REAL y, REAL z, uint type)
{
	Object* inside_obj = NULL;
	for (int n = 0; n < objects.size(); n++) {
		Object& obj = objects[n];

		bool found = false;
		REAL thickness = 3.0 / _gridRes;
		if (obj.shape == BOX) {
			if (x > obj.p[0].x && x < obj.p[1].x &&
				y > obj.p[0].y && y < obj.p[1].y &&
				z > obj.p[0].z && z < obj.p[1].z) {

				if (obj.type == WALL &&
					x > obj.p[0].x + thickness && x < obj.p[1].x - thickness &&
					y > obj.p[0].y + thickness && y < obj.p[1].y - thickness &&
					z > obj.p[0].z + thickness && z < obj.p[1].z - thickness) {
					// �� obj�� ��� ���� ���� �ȿ��� particle ���� X 
					inside_obj = NULL;
					break;
				}
				else {
					found = true;
				}
			}
		}
		else if (obj.shape == SPHERE) {
			REAL3 p = make_REAL3(x, y, z);
			REAL3 c = make_REAL3(obj.c.x, obj.c.y, obj.c.z);
		
			REAL len = Length(p - c);
			if (len < obj.r) {
				if (obj.type == WALL) {
					found = true;
					if (len < obj.r - thickness) {
						// �� obj�� ��� ���� ���� �ȿ��� particle ���� X 
						inside_obj = NULL;
						break;
					}
				}
				else if (obj.type == FLUID) {
					found = true;
				}
			}
		}

		if (found) {
			if (objects[n].type == type) {
				inside_obj = &objects[n]; // Found
				break;
			}
		}
	}

	if (inside_obj) {
		REAL _x = x + 0.01 * (inside_obj->type == FLUID) * 0.2 * ((rand() % 101) / 50.0 - 1.0) / _gridRes;
		REAL _y = y + 0.01 * (inside_obj->type == FLUID) * 0.2 * ((rand() % 101) / 50.0 - 1.0) / _gridRes;
		REAL _z = z + 0.01 * (inside_obj->type == FLUID) * 0.2 * ((rand() % 101) / 50.0 - 1.0) / _gridRes;

		REAL3 beforePos = make_REAL3(0.0, 0.0, 0.0);
		REAL3 curPos = make_REAL3(_x, _y, _z);
		REAL3 vel = make_REAL3(0.0, 0.0, 0.0);
		REAL3 normal = make_REAL3(0.0, 0.0, 0.0);
		REAL dens = 10.0;
		uint type = inside_obj->type;
		uint visible = inside_obj->visible;
		REAL mass = 1.0;
		BOOL flag = false;

		h_BeforePos.push_back(beforePos);
		h_CurPos.push_back(curPos);
		h_Vel.push_back(vel);
		h_Normal.push_back(normal);
		h_Dens.push_back(dens);
		h_Type.push_back(type);
		h_Visible.push_back(visible);
		h_Mass.push_back(mass);
		h_Flag.push_back(false);
	}
}

void FLIP3D_Cuda::ComputeWallParticleNormal_kernel()
{
	SetHashTable_kernel();

	ComputeWallParticleNormal_D << <divup(_numParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_CurPos(), d_Type(), d_Normal(), d_GridHash(), d_GridIdx(), d_CellStart(), d_CellEnd(), _numParticles, _gridRes);
	printf("Normal compute\n");
}

void FLIP3D_Cuda::ComputeParticleDensity_kernel()
{
	ComputeParticleDensity_D << <divup(_numParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_CurPos(), d_Type(), d_Dens(), d_Mass(), d_GridHash(), d_GridIdx(), d_CellStart(), d_CellEnd(), _gridRes, _numParticles, _dens, _maxDens, d_Flag());
}

void FLIP3D_Cuda::ComputeExternalForce_kernel(REAL3& gravity, REAL dt)
{
	CompExternlaForce_D << <divup(_numParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_CurPos(), d_Vel(), gravity, _externalForce, _numParticles, dt);
}

void FLIP3D_Cuda::SolvePICFLIP()
{
	//ResetCell_kernel();

	TrasnferToGrid_kernel();
	MarkWater_kernel();

	ComputeGridDensity_kernel();
	EnforceBoundary_kernel();
	ComputeDivergence_kernel();
	ComputeLevelSet_kernel();
	SolvePressureJacobi_kernel();
	ComputeVelocityWithPress_kernel();
	EnforceBoundary_kernel();
	ExtrapolateVelocity_kernel();

	SubtarctGrid_kernel();
	TrasnferToParticle_kernel();

	GridValueVisualize();
}

void FLIP3D_Cuda::ResetCell_kernel()
{
	ResetCell_D << <_grid->_cudaGridSize, _grid->_cudaBlockSize >> > (_grid->d_Volumes, _gridRes);
}

void FLIP3D_Cuda::TrasnferToGrid_kernel()
{
	TrasnferToGrid_D << <_grid->_cudaGridSize, _grid->_cudaBlockSize >> >
		(_grid->d_Volumes, d_CurPos(), d_Vel(), d_Type(), d_Mass(), d_GridHash(), d_GridIdx(), d_CellStart(), d_CellEnd(), _gridRes, _numParticles);
}

void FLIP3D_Cuda::MarkWater_kernel()
{
	MarkWater_D << <_grid->_cudaGridSize, _grid->_cudaBlockSize >> >
		(_grid->d_Volumes, d_CurPos(), d_Type(), d_Dens(), d_GridHash(), d_GridIdx(), d_CellStart(), d_CellEnd(), _dens, _gridRes);
}

void FLIP3D_Cuda::EnforceBoundary_kernel()
{
	EnforceBoundary_D << < _grid->_cudaGridSize, _grid->_cudaBlockSize >> > (_grid->d_Volumes, _gridRes);
}

void FLIP3D_Cuda::ComputeDivergence_kernel()
{
	ComputeDivergence_D << < _grid->_cudaGridSize, _grid->_cudaBlockSize >> > (_grid->d_Volumes, d_Dens(), d_GridHash(), d_GridIdx(), d_CellStart(), d_CellEnd(),_gridRes);
}

void FLIP3D_Cuda::ComputeLevelSet_kernel()
{
	ComputeLevelSet_D << < _grid->_cudaGridSize, _grid->_cudaBlockSize >> > (_grid->d_Volumes, d_CurPos(), d_Type(), d_Dens(), d_GridHash(), d_GridIdx(), d_CellStart(), d_CellEnd(), _dens, _gridRes);
}

void FLIP3D_Cuda::ComputeGridDensity_kernel()
{
	ComputeGridDensity_D << <_grid->_cudaGridSize, _grid->_cudaBlockSize >> >
		(_grid->d_Volumes, d_CurPos(), d_Type(), d_Mass(), d_GridHash(), d_GridIdx(), d_CellStart(), d_CellEnd(), _dens, _maxDens, _gridRes);
}

void FLIP3D_Cuda::SolvePressureJacobi_kernel()
{
	for (int i = 0; i < _iterations; i++)
	{
		SolvePressureJacobi_D << < _grid->_cudaGridSize, _grid->_cudaBlockSize >> > 
			(_grid->d_Volumes, _gridRes);
	}
}

void FLIP3D_Cuda::ComputeVelocityWithPress_kernel()
{
	ComputeVelocityWithPress_D << < _grid->_cudaGridSize, _grid->_cudaBlockSize >> > (_grid->d_Volumes, _gridRes);
}

void FLIP3D_Cuda::ExtrapolateVelocity_kernel()
{
	ExtrapolateVelocity_D << < _grid->_cudaGridSize, _grid->_cudaBlockSize >> > (_grid->d_Volumes, _gridRes);
}

void FLIP3D_Cuda::SubtarctGrid_kernel()
{
	SubtarctGrid_D << < _grid->_cudaGridSize, _grid->_cudaBlockSize >> > (_grid->d_Volumes, _gridRes);
}

void FLIP3D_Cuda::TrasnferToParticle_kernel()
{
	uint numThreads, numBlocks;
	ComputeGridSize(_numParticles, 128, numBlocks, numThreads);
	TrasnferToParticle_D << <numBlocks, numThreads >> > (_grid->d_Volumes, _gridRes, d_CurPos(), d_Vel(), _numParticles);
}

void FLIP3D_Cuda::AdvectParticle_kernel(REAL dt)
{
	AdvecParticle_D << < divup(_numParticles, BLOCK_SIZE), BLOCK_SIZE >> > 
		(_grid->d_Volumes, d_BeforePos(), d_CurPos(), d_Vel(), d_Type(), _gridRes, _numParticles, dt);

	SetHashTable_kernel();

	ConstraintOuterWall_D << < divup(_numParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_CurPos(), d_Vel(), d_Normal(), d_Type(), d_GridHash(), d_GridIdx(), d_CellStart(), d_CellEnd(), _numParticles, _gridRes, _dens);
}

void FLIP3D_Cuda::Correct_kernel(REAL dt)
{
	SetHashTable_kernel();

	uint r1 = rand();
	uint r2 = rand();
	uint r3 = rand();
	Correct_D << < divup(_numParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_CurPos(), d_Vel(), d_Normal(), d_Mass(), d_Type(), d_GridHash(), d_GridIdx(), d_CellStart(), d_CellEnd(), _gridRes, _numParticles, dt, _dens / _gridRes, r1, r2, r3);
}
void FLIP3D_Cuda::SetHashTable_kernel(void)
{
	CalculateHash_kernel();
	SortParticle_kernel();
	FindCellStart_kernel();
}

void FLIP3D_Cuda::CalculateHash_kernel(void)
{
	CalculateHash_D << <divup(_numParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_GridHash(), d_GridIdx(), d_CurPos(), _gridRes, _numParticles);
}

void FLIP3D_Cuda::SortParticle_kernel(void)
{
	thrust::sort_by_key(thrust::device_ptr<uint>(d_GridHash()),
		thrust::device_ptr<uint>(d_GridHash() + _numParticles),
		thrust::device_ptr<uint>(d_GridIdx()));
}

void FLIP3D_Cuda::FindCellStart_kernel(void)
{
	uint numThreads, numBlocks;
	ComputeGridSize(_numParticles, 128, numBlocks, numThreads);

	uint smemSize = sizeof(uint) * (numThreads + 1);
	FindCellStart_D << <numBlocks, numThreads, smemSize >> >
		(d_GridHash(), d_CellStart(), d_CellEnd(), _numParticles);
}

void FLIP3D_Cuda::InitDeviceMem(void)
{
	d_GridHash.resize(_numParticles);			d_GridHash.memset(0);
	d_GridIdx.resize(_numParticles);			d_GridIdx.memset(0);
	d_CellStart.resize(_gridRes * _gridRes * _gridRes);			d_CellStart.memset(0);
	d_CellEnd.resize(_gridRes * _gridRes * _gridRes);			d_CellEnd.memset(0);

	d_BeforePos.resize(_numParticles);			d_BeforePos.memset(0);
	d_CurPos.resize(_numParticles);			d_CurPos.memset(0);
	d_Vel.resize(_numParticles);			d_Vel.memset(0);
	d_Normal.resize(_numParticles);			d_Normal.memset(0);
	d_Type.resize(_numParticles);			d_Type.memset(0);
	d_Visible.resize(_numParticles);		d_Visible.memset(0);
	d_Remove.resize(_numParticles);			d_Remove.memset(0);
	d_Mass.resize(_numParticles);			d_Mass.memset(0);
	d_Dens.resize(_numParticles);			d_Dens.memset(0);

	d_Flag.resize(_numParticles);			d_Flag.memset(0);

	d_gridPos.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1));		d_gridPos.memset(0);
	d_gridVel.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1));		d_gridVel.memset(0);
	d_gridPress.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1));		d_gridPress.memset(0);
	d_gridDens.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1));		d_gridDens.memset(0);
	d_gridLevelSet.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1));		d_gridLevelSet.memset(0);
	d_gridDiv.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1));		d_gridDiv.memset(0);
	d_gridContent.resize((_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1));		d_gridContent.memset(0);
	printf("Size: %d\n", (_gridRes + 1) * (_gridRes + 1) * (_gridRes + 1));
}

void FLIP3D_Cuda::FreeDeviceMem(void)
{
	d_GridHash.free();
	d_GridIdx.free();
	d_CellStart.free();
	d_CellEnd.free();

	d_BeforePos.free();
	d_CurPos.free();
	d_Vel.free();
	d_Normal.free();
	d_Type.free();
	d_Visible.free();
	d_Remove.free();
	d_Mass.free();
	d_Dens.free();

	d_Flag.free();
	d_gridPos.free();
	d_gridVel.free();
	d_gridPress.free();
	d_gridDens.free();
	d_gridLevelSet.free();
	d_gridDiv.free();
	d_gridContent.free();

}

void FLIP3D_Cuda::CopyToDevice(void)
{
	d_BeforePos.copyFromHost(h_BeforePos);
	d_CurPos.copyFromHost(h_CurPos);
	d_Vel.copyFromHost(h_Vel);
	d_Normal.copyFromHost(h_Normal);
	d_Type.copyFromHost(h_Type);
	d_Visible.copyFromDevice(h_Visible);
	d_Remove.copyFromHost(h_Remove);
	d_Mass.copyFromHost(h_Mass);
	d_Dens.copyFromHost(h_Dens);

	d_Flag.copyFromHost(h_Flag);
	d_gridPos.copyFromHost(h_gridPos);
	d_gridVel.copyFromHost(h_gridVel);
	d_gridPress.copyFromHost(h_gridPress);
	d_gridDens.copyFromHost(h_gridDens);
	d_gridLevelSet.copyFromHost(h_gridLevelSet);
	d_gridDiv.copyFromHost(h_gridDiv);
	d_gridContent.copyFromHost(h_gridContent);
}

void FLIP3D_Cuda::CopyToHost(void)
{
	d_BeforePos.copyToHost(h_BeforePos);
	d_CurPos.copyToHost(h_CurPos);
	d_Vel.copyToHost(h_Vel);
	d_Normal.copyToHost(h_Normal);
	d_Type.copyToHost(h_Type);
	d_Visible.copyToHost(h_Visible);
	d_Remove.copyToHost(h_Remove);
	d_Mass.copyToHost(h_Mass);
	d_Dens.copyToHost(h_Dens);

	d_Flag.copyToHost(h_Flag);
	d_gridPos.copyToHost(h_gridPos);
	d_gridVel.copyToHost(h_gridVel);
	d_gridPress.copyToHost(h_gridPress);
	d_gridDens.copyToHost(h_gridDens);
	d_gridLevelSet.copyToHost(h_gridLevelSet);
	d_gridDiv.copyToHost(h_gridDiv);
	d_gridContent.copyToHost(h_gridContent);
}

void FLIP3D_Cuda::GridValueVisualize(void)
{
	GridVisualize_D << < _grid->_cudaGridSize, _grid->_cudaBlockSize >> > (_grid->d_Volumes, _gridRes, d_gridPos(), d_gridVel(), d_gridPress(), d_gridDens(), d_gridLevelSet(), d_gridDiv(), d_gridContent());
}

void FLIP3D_Cuda::draw(void)
{
	int cnt = 0;
	glPushMatrix();
	glDisable(GL_LIGHTING);
	glPointSize(1.0);
	for (uint i = 0u; i < _numParticles; i++)
	{
		REAL3 position = h_CurPos[i];
		REAL3 velocity = h_Vel[i];
		REAL3 normal = h_Normal[i];
		REAL density = h_Dens[i];
		uint type = h_Type[i];
		BOOL flag = h_Flag[i];

		//if (h_Flag[i])
		//	glColor3f(1.0f, 0.0f, 0.0f);
		//else
		//{
		//	//continue;
		//	glColor3f(0.0f, 0.0f, 1.0f);
		//}

		if (type == WALL ) {
			continue;
			glColor3f(1.0f, 1.0f, 1.0f);
		}
		else
			glColor3f(0.0f, 1.0f, 1.0f);
		//glColor3f(1.0, 1.0, 1.0);
		cnt++;
		//////////cout << h_Dens[i] << endl;
		//REAL3 color = ScalarToColor(density);
		//glColor3f(color.x, color.y, color.z);

		glBegin(GL_POINTS);
		glVertex3d(position.x, position.y, position.z);
		glEnd();

		//glColor3f(1.0f, 1.0f, 1.0f);
		//glLineWidth(1.0f);
		//glBegin(GL_LINES);
		//float c = 0.2f;
		//glVertex3d(position.x, position.y, position.z);
		//glVertex3d(position.x + velocity.x * c, position.y + velocity.y * c, position.z + velocity.z * c);
		//glEnd();

		//glColor3f(1.0f, 1.0f, 1.0f);
		//glLineWidth(1.0f);
		//glBegin(GL_LINES);
		//float c = 0.2f;
		//glVertex3d(position.x, position.y, position.z);
		//glVertex3d(position.x + normal.x * c, position.y + normal.y * c, position.z + normal.z * c);
		//glEnd();
	}
	//printf("cnt: %d\n", cnt);

	for (uint i = 0u; i < _gridRes * _gridRes * _gridRes; i++)
	{
		REAL3 position = h_gridPos[i];
		REAL3 velocity = h_gridVel[i];
		REAL pressure = h_gridPress[i];
		REAL density = h_gridDens[i];
		REAL levelSet = h_gridLevelSet[i];
		REAL divergence = h_gridDiv[i];
		uint content = h_gridContent[i];

#if VEL

		glColor3f(1.0f, 1.0f, 1.0f);
		glLineWidth(1.0f);
		glBegin(GL_LINES);
		float c = 0.2f;
		glVertex3d(position.x, position.y, position.z);
		glVertex3d(position.x + velocity.x * c, position.y + velocity.y * c, position.z + velocity.z * c);
		glEnd();
#endif

#if PRESS
		//Visualize Pressure
		if (pressure == 0 )
			continue;
		REAL3 color = ScalarToColor(pressure * 10);
		glColor3f(color.x, color.y, color.z);

		glPointSize(15.0);
		glBegin(GL_POINTS);
		glVertex3d(position.x, position.y, position.z);
		glEnd();
#endif

#if DENSITY
		////Visualize Dens
		if (density == 0 || content == CONTENT_WALL || content == CONTENT_AIR)
			continue;
		REAL3 color = ScalarToColor(density);
		glColor3f(color.x, color.y, color.z);

		glPointSize(15.0);
		glBegin(GL_POINTS);
		glVertex3d(position.x, position.y, position.z);
		glEnd();
#endif

#if LEVEL
		//Visualize Level
		if (content == CONTENT_WALL )
			continue;
		REAL3 color = ScalarToColor(abs(levelSet) * 0.1);
		glColor3f(color.x, color.y, color.z);

		if (content == CONTENT_FLUID)
			glPointSize(15.0);
		else if (content == CONTENT_AIR) {
			continue;
			glPointSize(2.0);
		}
		glBegin(GL_POINTS);
		glVertex3d(position.x, position.y, position.z);
		glEnd();
#endif

#if DIV
		//Visualize Level
		if (content == CONTENT_WALL)
			continue;
		REAL3 color = ScalarToColor(abs(divergence) * 0.1);
		glColor3f(color.x, color.y, color.z);

		if (content == CONTENT_FLUID)
			glPointSize(15.0);
		else if (content == CONTENT_AIR)
			glPointSize(2.0);
		glBegin(GL_POINTS);
		glVertex3d(position.x, position.y, position.z);
		glEnd();
#endif

#if CONTENT
		//////Visualize Content
		if (content == CONTENT_FLUID) {
			//continue;
			glColor3f(0, 0, 1);
			glPointSize(15.0);

		}
		else if (content == CONTENT_AIR) {
			//continue;
			glColor3f(0, 1, 0);
			glPointSize(2.0);

		}
		else if (content == CONTENT_WALL) {
			//continue;
			glColor3f(1, 1, 1);
			glPointSize(1.0);
		}

		glBegin(GL_POINTS);
		glVertex3d(position.x, position.y, position.z);
		glEnd();

#endif
	}
	glPointSize(1.0);
	glEnable(GL_LIGHTING);
	glPopMatrix();
}

REAL3 FLIP3D_Cuda::ScalarToColor(double val)
{
	double fColorMap[5][3] = { { 0,0,1 },{ 0,1,1 },{ 0,1,0 },{ 1,1,0 },{ 1,0,0 } };   //Red->Blue
	double v = val;
	if (val > 1.0) v = 1.0; if (val < 0.0) v = 0.0; v *= 4.0;
	int low = (int)floor(v), high = (int)ceil(v);
	double t = v - low;
	REAL x = (fColorMap[low][0]) * (1 - t) + (fColorMap[high][0]) * t;
	REAL y = (fColorMap[low][1]) * (1 - t) + (fColorMap[high][1]) * t;
	REAL z = (fColorMap[low][2]) * (1 - t) + (fColorMap[high][2]) * t;
	REAL3 color = make_REAL3(x, y, z);
	return color;
}