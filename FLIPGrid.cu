#include "hip/hip_runtime.h"
#include "FLIPGrid.cuh"

FLIPGRID::FLIPGRID()
{

}

FLIPGRID::FLIPGRID(uint res, REAL cellPhysicalSize):
	_gridRes(res), _gridCellCount((res+1)* (res + 1)* (res + 1)), _cellPhysicalSize(cellPhysicalSize), _physicalSize(res * cellPhysicalSize)
{
	_cudaGridSize = dim3(divup(_gridRes + 1, _cudaBlockSize.x), divup(_gridRes + 1, _cudaBlockSize.y), divup(_gridRes + 1, _cudaBlockSize.z));
	printf("cudaGridSize: %d %d %d\n", _cudaGridSize.x, _cudaGridSize.y, _cudaGridSize.z);

	d_Volumes.content = createField3D<int>(_gridRes + 1, _gridRes + 1, _gridRes + 1, _cudaGridSize, _cudaBlockSize, CONTENT_AIR, false);
	d_Volumes.pressure = createField3D<float>(_gridRes, _gridRes, _gridRes, _cudaGridSize, _cudaBlockSize, 0.f, false);
	d_Volumes.fluidIndex = createField3D<int>(_gridRes, _gridRes, _gridRes, _cudaGridSize, _cudaBlockSize, 0, false);
	d_Volumes.divergence = createField3D<float>(_gridRes, _gridRes, _gridRes, _cudaGridSize, _cudaBlockSize, 0.f, false);
	d_Volumes.particleCount = createField3D<int>(_gridRes, _gridRes, _gridRes, _cudaGridSize, _cudaBlockSize, 0, false);

	d_Volumes.velocityAccumWeight = createField3D<float4>(_gridRes, _gridRes, _gridRes, _cudaGridSize, _cudaBlockSize, make_float4(0, 0, 0, 0), false);
	d_Volumes.hasVelocity = createField3D<int4>(_gridRes, _gridRes, _gridRes, _cudaGridSize, _cudaBlockSize, make_int4(0, 0, 0, 0), false);


	d_Volumes.velocity = createField3D<float4>(_gridRes + 1, _gridRes + 1, _gridRes + 1, _cudaGridSize, _cudaBlockSize, make_float4(0, 0, 0, 0), true);

	d_Volumes.newVelocity = createField3D<float4>(_gridRes + 1, _gridRes + 1, _gridRes + 1, _cudaGridSize, _cudaBlockSize, make_float4(0, 0, 0, 0), true);


	d_Volumes.volumeFractions = createField3D<float4>(_gridRes, _gridRes, _gridRes, _cudaGridSize, _cudaBlockSize, make_float4(0, 0, 0, 0), false);
	d_Volumes.newVolumeFractions = createField3D<float4>(_gridRes, _gridRes, _gridRes, _cudaGridSize, _cudaBlockSize, make_float4(0, 0, 0, 0), false);

	d_Volumes.density = createField3D<float>(_gridRes, _gridRes, _gridRes, _cudaGridSize, _cudaBlockSize, 0.f, false);

	Init();
}

FLIPGRID::~FLIPGRID()
{
	releaseField3D(d_Volumes.content);
	releaseField3D(d_Volumes.pressure);
	releaseField3D(d_Volumes.fluidIndex);
	releaseField3D(d_Volumes.divergence);
	releaseField3D(d_Volumes.particleCount);

	releaseField3D(d_Volumes.velocityAccumWeight);
	releaseField3D(d_Volumes.hasVelocity);

	releaseField3D(d_Volumes.velocity);
	releaseField3D(d_Volumes.newVelocity);

	releaseField3D(d_Volumes.volumeFractions);
	releaseField3D(d_Volumes.newVolumeFractions);
}


void FLIPGRID::Init(void)
{
}

void FLIPGRID::InitDeviceMem(void)
{

}

void FLIPGRID::FreeDeviceMem(void)
{

}

void FLIPGRID::CopyToDevice(void)
{

}

void FLIPGRID::CopyToHost(void)
{

}