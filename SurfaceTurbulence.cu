#include "hip/hip_runtime.h"
#include "SurfaceTurbulence.cuh"

SurfaceTurbulence::SurfaceTurbulence()
{

}

SurfaceTurbulence::SurfaceTurbulence(FLIP3D_Cuda* fluid, uint gridRes) {
	_fluid = fluid;

	_coarseScaleLen = 1.0 / gridRes;
	_baseRes = gridRes;
	_fineScaleLen = PI * (_coarseScaleLen + (_coarseScaleLen / 2.0)) / SURFACE_DENSITY;
	_hashGridRes = _baseRes * 4;

	_outerRadius = _coarseScaleLen;
	_innerRadius = _outerRadius / 2.0;

	_waveSeedingCurvatureThresholdMinimum = _coarseScaleLen * 0.005; //��� �Ӱ谪 (���� �ʿ�)
	_waveSeedingCurvatureThresholdMaximum = _coarseScaleLen * 0.077;

	InitHostMem();
	InitDeviceMem();
	CopyToDevice();

	Initialize_kernel();
	printf("Coarse Scale Length: %f\n", _coarseScaleLen);
	printf("Fine Scale Length: %f\n", _fineScaleLen);

	CopyToHost();
	printf("Initialize coarse-particles number is %d\n", _fluid->_numParticles);
	printf("Initialize fine-particles number is %d\n", _numFineParticles);
}

SurfaceTurbulence:: ~SurfaceTurbulence()
{
	FreeDeviceMem();
}

void SurfaceTurbulence::ThrustScanWrapper_kernel(uint* output, uint* input, uint numElements)
{
	thrust::exclusive_scan(thrust::device_ptr<uint>(input),
		thrust::device_ptr<uint>(input + (_fluid->_numParticles * PER_PARTICLE)),
		thrust::device_ptr<uint>(output));
}

void SurfaceTurbulence::Initialize_kernel()
{
	_fluid->SetHashTable_kernel();

	Initialize_D << <divup(_fluid->_numParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(_fluid->d_CurPos(), _fluid->d_Type(), this->d_Pos(), this->d_ParticleGridIndex(), _fluid->_numParticles, _fluid->d_GridIdx(), _fluid->d_CellStart(), _fluid->d_CellEnd(), _baseRes,
			_fineScaleLen, _outerRadius, _innerRadius);

	thrust::sort_by_key(thrust::device_ptr<uint>(d_ParticleGridIndex()),
		thrust::device_ptr<uint>(d_ParticleGridIndex() + (_fluid->_numParticles * PER_PARTICLE)),
		thrust::device_ptr<REAL3>(d_Pos()));

	StateCheck_D << <divup(_fluid->_numParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(this->d_Pos(), this->d_ParticleGridIndex(), this->d_StateData(), _fluid->_numParticles);

	ThrustScanWrapper_kernel(this->d_StateData(), this->d_StateData(), (_fluid->_numParticles * PER_PARTICLE));

	CUDA_CHECK(hipMemcpy((void*)&_numFineParticles, (void*)(this->d_StateData() + (_fluid->_numParticles * PER_PARTICLE) - 1), sizeof(uint), hipMemcpyDeviceToHost));
}

void SurfaceTurbulence::Advection_kernel(void)
{
	_fluid->SetHashTable_kernel();
	REAL r = 2.0 * _coarseScaleLen;

	ComputeCoarseDens_D << <divup(_fluid->_numParticles, BLOCK_SIZE), BLOCK_SIZE >> >
	(r, _fluid->d_CurPos(), _fluid->d_Type(), _fluid->d_KernelDens(), _fluid->d_GridIdx(), _fluid->d_CellStart(), _fluid->d_CellEnd(), 
		_baseRes, _fluid->_numParticles);
 
	Advection_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(this->d_Pos(), _fluid->d_CurPos(), _fluid->d_BeforePos(), _fluid->d_Type(), _fluid->d_KernelDens(), _fluid->d_GridIdx(), _fluid->d_CellStart(), _fluid->d_CellEnd(), 
			_numFineParticles, _coarseScaleLen, _baseRes, d_Flag());
}

void SurfaceTurbulence::SurfaceConstraint_kernel(void)
{
	SurfaceConstraint_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(this->d_Pos(), _fluid->d_CurPos(), _fluid->d_GridIdx(), _fluid->d_CellStart(), _fluid->d_CellEnd(), _outerRadius, _innerRadius, _baseRes, _numFineParticles, d_SurfaceNormal());
	
}

void SurfaceTurbulence::ComputeSurfaceNormal_kernel(void)
{
	REAL r = _coarseScaleLen;
	ComputeCoarseDens_D << <divup(_fluid->_numParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(r, _fluid->d_CurPos(), _fluid->d_Type(), _fluid->d_KernelDens(), _fluid->d_GridIdx(), _fluid->d_CellStart(), _fluid->d_CellEnd(),
			_baseRes, _fluid->_numParticles);

	ComputeFineDens_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(r, d_Pos(), d_KernelDens(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles);

	ComputeFineNeighborWeightSum_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(r, d_Pos(), d_KernelDens(), d_NeighborWeightSum(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles);

	ComputeSurfaceNormal_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(_fluid->d_CurPos(), _fluid->d_GridIdx(), _fluid->d_CellStart(), _fluid->d_CellEnd(), _baseRes,
			d_Pos(), d_KernelDens(), d_NeighborWeightSum(), d_TempNormal(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles,
			_outerRadius, _innerRadius);

	SmoothNormal_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_Pos(), d_KernelDens(), d_NeighborWeightSum(), d_TempNormal(), d_SurfaceNormal(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles, _baseRes);
}

void SurfaceTurbulence::NormalRegularization_kernel(void)
{
	REAL r = _coarseScaleLen;

	ComputeFineDens_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(r, d_Pos(), d_KernelDens(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles);

	ComputeFineNeighborWeightSum_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(r, d_Pos(), d_KernelDens(), d_NeighborWeightSum(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles);

	NormalRegularization_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_Pos(), d_TempPos(), d_SurfaceNormal(), d_KernelDens(), d_NeighborWeightSum(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles, _baseRes);
}

void SurfaceTurbulence::TangentRegularization_kernel(void)
{
	REAL r = 3.0 * _fineScaleLen;

	ComputeFineDens_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(r, d_Pos(), d_KernelDens(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles);

	ComputeFineNeighborWeightSum_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(r, d_Pos(), d_KernelDens(), d_NeighborWeightSum(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles);

	TangentRegularization_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_Pos(), d_TempPos(), d_SurfaceNormal(), d_KernelDens(), d_NeighborWeightSum(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles);

}

void SurfaceTurbulence::Regularization_kernel(void)
{

	ComputeSurfaceNormal_kernel();

	CopyToTempPos_D << < divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_Pos(), d_TempPos(), _numFineParticles);

	NormalRegularization_kernel();
	TangentRegularization_kernel();

	CopyToPos_D << < divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_Pos(), d_TempPos(), _numFineParticles);
}

void SurfaceTurbulence::InsertFineParticles(void)
{
	REAL tangentRadius = 3.0 * _fineScaleLen;

	//Normal ���
	ComputeSurfaceNormal_kernel();

	//Tangent Regularization�� ���� ����ġ �е� ����
	ComputeFineDens_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(tangentRadius, d_Pos(), d_KernelDens(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles);

	ComputeFineNeighborWeightSum_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(tangentRadius, d_Pos(), d_KernelDens(), d_NeighborWeightSum(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles);

	InsertFineParticles_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_ParticleGridIndex(), d_Pos(), d_SurfaceNormal(), d_KernelDens(), d_NeighborWeightSum(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles, _fluid->_numParticles);

	thrust::sort_by_key(thrust::device_ptr<uint>(d_ParticleGridIndex()),
		thrust::device_ptr<uint>(d_ParticleGridIndex() + (_fluid->_numParticles * PER_PARTICLE)),
		thrust::device_ptr<REAL3>(d_Pos()));

	StateCheck_D << <divup(_fluid->_numParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_Pos(), d_ParticleGridIndex(), d_StateData(), _fluid->_numParticles);

	ThrustScanWrapper_kernel(d_StateData(), d_StateData(), (_fluid->_numParticles * PER_PARTICLE));

	CUDA_CHECK(hipMemcpy((void*)&_numFineParticles, (void*)(this->d_StateData() + (_fluid->_numParticles * PER_PARTICLE) - 1), sizeof(uint), hipMemcpyDeviceToHost));

}

void SurfaceTurbulence::DeleteFineParticles(void)
{
	DeleteFineParticles_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_ParticleGridIndex(), d_Pos(), d_SurfaceNormal(), d_KernelDens(), d_NeighborWeightSum(), d_GridIdx(), d_CellStart(), d_CellEnd(), _hashGridRes, _numFineParticles, _fluid->_numParticles);

	AdvectionDeleteFineParticles_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_ParticleGridIndex(), d_Pos(), _fluid->d_CurPos(), _fluid->d_Type(), _fluid->d_GridIdx(), _fluid->d_CellStart(), _fluid->d_CellEnd(), _numFineParticles, _fluid->_numParticles, _baseRes);

	ConstraintDeleteFineParticles_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_ParticleGridIndex(), d_Pos(), _fluid->d_CurPos(), _fluid->d_GridIdx(), _fluid->d_CellStart(), _fluid->d_CellEnd(), _numFineParticles, _fluid->_numParticles, _baseRes, _outerRadius, _innerRadius);

	thrust::sort_by_key(thrust::device_ptr<uint>(d_ParticleGridIndex()),
		thrust::device_ptr<uint>(d_ParticleGridIndex() + (_fluid->_numParticles * PER_PARTICLE)),
		thrust::device_ptr<REAL3>(d_Pos()));

	StateCheck_D << <divup(_fluid->_numParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_Pos(), d_ParticleGridIndex(), d_StateData(), _fluid->_numParticles);

	ThrustScanWrapper_kernel(d_StateData(), d_StateData(), (_fluid->_numParticles * PER_PARTICLE));

	CUDA_CHECK(hipMemcpy((void*)&_numFineParticles, (void*)(this->d_StateData() + (_fluid->_numParticles * PER_PARTICLE) - 1), sizeof(uint), hipMemcpyDeviceToHost));
}

void SurfaceTurbulence::SurfaceMaintenance(void)
{
	SurfaceConstraint_kernel();

	SetHashTable_kernel();
	Regularization_kernel();

	SetHashTable_kernel();
	InsertFineParticles();
	DeleteFineParticles();
}

void SurfaceTurbulence::SetHashTable_kernel(void)
{
	CalculateHash_kernel();
	SortParticle_kernel();
	FindCellStart_kernel();
}

void SurfaceTurbulence::CalculateHash_kernel(void)
{
	CalculateHash_D << <divup(_numFineParticles, BLOCK_SIZE), BLOCK_SIZE >> >
		(d_GridHash(), d_GridIdx(), d_Pos(), _hashGridRes, _numFineParticles);
}

void SurfaceTurbulence::SortParticle_kernel(void)
{
	thrust::sort_by_key(thrust::device_ptr<uint>(d_GridHash()),
		thrust::device_ptr<uint>(d_GridHash() + _numFineParticles),
		thrust::device_ptr<uint>(d_GridIdx()));
}

void SurfaceTurbulence::FindCellStart_kernel(void)
{
	uint numThreads, numBlocks;
	ComputeGridSize(_numFineParticles, 128, numBlocks, numThreads);

	uint smemSize = sizeof(uint) * (numThreads + 1);
	FindCellStart_D << <numBlocks, numThreads, smemSize >> >
		(d_GridHash(), d_CellStart(), d_CellEnd(), _numFineParticles);
}

void SurfaceTurbulence::InitHostMem(void)
{
	//Surface Maintenance
	h_Pos.resize(_fluid->_numParticles * PER_PARTICLE);
	h_Vel.resize(_fluid->_numParticles * PER_PARTICLE);
	h_SurfaceNormal.resize(_fluid->_numParticles * PER_PARTICLE);
	h_TempNormal.resize(_fluid->_numParticles * PER_PARTICLE);
	h_TempPos.resize(_fluid->_numParticles * PER_PARTICLE);
	h_Tangent.resize(_fluid->_numParticles * PER_PARTICLE);
	h_KernelDens.resize(_fluid->_numParticles * PER_PARTICLE);
	h_NeighborWeightSum.resize(_fluid->_numParticles * PER_PARTICLE);
	h_Flag.resize(_fluid->_numParticles * PER_PARTICLE);

	////Wave Simulation
	//h_Curvature.resize(_fluid->_numParticles * PER_PARTICLE);
	//h_TempCurvature.resize(_fluid->_numParticles * PER_PARTICLE);
	//h_WaveH.resize(_fluid->_numParticles * PER_PARTICLE);
	//h_WaveDtH.resize(_fluid->_numParticles * PER_PARTICLE);
	//h_Seed.resize(_fluid->_numParticles * PER_PARTICLE);
	//h_WaveSeedAmp.resize(_fluid->_numParticles * PER_PARTICLE);
	//h_Laplacian.resize(_fluid->_numParticles * PER_PARTICLE);
	//h_WaveNormal.resize(_fluid->_numParticles * PER_PARTICLE);

}

void SurfaceTurbulence::InitDeviceMem()
{
	//Initialize
	d_ParticleGridIndex.resize(_fluid->_numParticles * PER_PARTICLE);	d_ParticleGridIndex.memset(0);
	d_StateData.resize(_fluid->_numParticles * PER_PARTICLE);			d_StateData.memset(0);

	//Surface Maintenance
	d_Pos.resize(_fluid->_numParticles * PER_PARTICLE);					d_Pos.memset(0);
	d_Vel.resize(_fluid->_numParticles * PER_PARTICLE);					d_Vel.memset(0);
	d_SurfaceNormal.resize(_fluid->_numParticles * PER_PARTICLE);		d_SurfaceNormal.memset(0);
	d_TempNormal.resize(_fluid->_numParticles * PER_PARTICLE);			d_TempNormal.memset(0);
	d_TempPos.resize(_fluid->_numParticles * PER_PARTICLE);				d_TempPos.memset(0);
	d_Tangent.resize(_fluid->_numParticles * PER_PARTICLE);				d_Tangent.memset(0);
	d_KernelDens.resize(_fluid->_numParticles * PER_PARTICLE);			d_KernelDens.memset(0);
	d_NeighborWeightSum.resize(_fluid->_numParticles * PER_PARTICLE);	d_NeighborWeightSum.memset(0);
	d_Flag.resize(_fluid->_numParticles * PER_PARTICLE);				d_Flag.memset(0);

	////Wave Simulation
	//d_Curvature.resize(_fluid->_numParticles * PER_PARTICLE);			d_Curvature.memset(0);
	//d_TempCurvature.resize(_fluid->_numParticles * PER_PARTICLE);		d_TempCurvature.memset(0);
	//d_WaveH.resize(_fluid->_numParticles * PER_PARTICLE);				d_WaveH.memset(0);
	//d_WaveDtH.resize(_fluid->_numParticles * PER_PARTICLE);			d_WaveDtH.memset(0);
	//d_Seed.resize(_fluid->_numParticles * PER_PARTICLE);				d_Seed.memset(0);
	//d_WaveSeedAmp.resize(_fluid->_numParticles * PER_PARTICLE);		d_WaveSeedAmp.memset(0);
	//d_Laplacian.resize(_fluid->_numParticles * PER_PARTICLE);			d_Laplacian.memset(0);
	//d_WaveNormal.resize(_fluid->_numParticles * PER_PARTICLE);			d_WaveNormal.memset(0);

	//Hash
	d_GridHash.resize(_fluid->_numParticles * PER_PARTICLE);			d_GridHash.memset(0);
	d_GridIdx.resize(_fluid->_numParticles * PER_PARTICLE);			d_GridIdx.memset(0);
	d_CellStart.resize(_hashGridRes * _hashGridRes * _hashGridRes);			d_CellStart.memset(0);
	d_CellEnd.resize(_hashGridRes * _hashGridRes * _hashGridRes);			d_CellEnd.memset(0);
}

void SurfaceTurbulence::FreeDeviceMem()
{
	//Initialize
	d_ParticleGridIndex.free();
	d_StateData.free();

	//Surface Maintenance
	d_Pos.free();
	d_Vel.free();
	d_SurfaceNormal.free();
	d_TempNormal.free();
	d_TempPos.free();
	d_Tangent.free();
	d_KernelDens.free();
	d_NeighborWeightSum.free();
	d_Flag.free();

	////Wave Simulation
	//d_Curvature.free();
	//d_TempCurvature.free();
	//d_WaveH.free();
	//d_WaveDtH.free();
	//d_Seed.free();
	//d_WaveSeedAmp.free();
	//d_Laplacian.free();
	//d_WaveNormal.free();

	//Hash
	d_GridHash.free();
	d_GridIdx.free();
	d_CellStart.free();
	d_CellEnd.free();
}

void SurfaceTurbulence::CopyToDevice()
{
	//Surface Maintenance
	d_Pos.copyFromHost(h_Pos);
	d_Vel.copyFromHost(h_Vel);
	d_SurfaceNormal.copyFromHost(h_SurfaceNormal);
	d_TempNormal.copyFromHost(h_TempNormal);
	d_TempPos.copyFromHost(h_TempPos);
	d_Tangent.copyFromHost(h_Tangent);
	d_KernelDens.copyFromHost(h_KernelDens);
	d_NeighborWeightSum.copyFromHost(h_NeighborWeightSum);
	d_Flag.copyFromHost(h_Flag);

	////Wave Simulation
	//d_Curvature.copyFromHost(h_Curvature);
	//d_TempCurvature.copyFromHost(h_TempCurvature);
	//d_WaveH.copyFromHost(h_WaveH);
	//d_WaveDtH.copyFromHost(h_WaveDtH);
	//d_Seed.copyFromHost(h_Seed);
	//d_WaveSeedAmp.copyFromHost(h_WaveSeedAmp);
	//d_Laplacian.copyFromHost(h_Laplacian);
	//d_WaveNormal.copyFromHost(h_WaveNormal);

}

void SurfaceTurbulence::CopyToHost()
{
	//Surface Maintenance
	d_Pos.copyToHost(h_Pos);
	d_Vel.copyToHost(h_Vel);
	d_SurfaceNormal.copyToHost(h_SurfaceNormal);
	d_TempNormal.copyToHost(h_TempNormal);
	d_TempPos.copyToHost(h_TempPos);
	d_Tangent.copyToHost(h_Tangent);
	d_KernelDens.copyToHost(h_KernelDens);
	d_NeighborWeightSum.copyToHost(h_NeighborWeightSum);
	d_Flag.copyToHost(h_Flag);
	////Wave Simulation
	//d_Curvature.copyToHost(h_Curvature);
	//d_TempCurvature.copyToHost(h_TempCurvature);
	//d_WaveH.copyToHost(h_WaveH);
	//d_WaveDtH.copyToHost(h_WaveDtH);
	//d_Seed.copyToHost(h_Seed);
	//d_WaveSeedAmp.copyToHost(h_WaveSeedAmp);
	//d_Laplacian.copyToHost(h_Laplacian);
	//d_WaveNormal.copyToHost(h_WaveNormal);

}

void SurfaceTurbulence::draw(void)
{
	glPushMatrix();
	glDisable(GL_LIGHTING);
	glPointSize(2.0);
	glLineWidth(1.0);
	for (uint i = 0u; i < _numFineParticles; i++)
	{
		REAL3 position = h_Pos[i];
		//REAL3 surfaceNormal = h_SurfaceNormal[i];
		REAL3 surfaceNormal = h_TempNormal[i];
		BOOL flag = h_Flag[i];

		//////Draw normal
		//glColor3f(1.0f, 1.0f, 1.0f);
		//double scale = 0.03;
		//glBegin(GL_LINES);
		//glVertex3d(position.x, position.y, position.z);
		//glVertex3d(position.x + surfaceNormal.x * scale, position.y + surfaceNormal.y * scale, position.z + surfaceNormal.z * scale);
		//glEnd();

		////general visualize
		glColor3f(1.0f, 0.0f, 0.0f);

		//if (flag) {
		//	glColor3f(1.0f, 0.0f, 0.0f);
		//}
		//else
		//	glColor3f(1.0f, 1.0f, 1.0f);
		glBegin(GL_POINTS);
		glVertex3d(position.x, position.y, position.z);
		glEnd();
	}
	//printf("NUM:::: %d\n", cnt);
	glPointSize(1.0);
	glEnable(GL_LIGHTING);
	glPopMatrix();
}

REAL3 SurfaceTurbulence::ScalarToColor(double val)
{
	double fColorMap[5][3] = { { 0,0,1 },{ 0,1,1 },{ 0,1,0 },{ 1,1,0 },{ 1,0,0 } };   //Red->Blue
	double v = val;
	if (val > 1.0) v = 1.0; if (val < 0.0) v = 0.0; v *= 4.0;
	int low = (int)floor(v), high = (int)ceil(v);
	double t = v - low;
	REAL x = (fColorMap[low][0]) * (1 - t) + (fColorMap[high][0]) * t;
	REAL y = (fColorMap[low][1]) * (1 - t) + (fColorMap[high][1]) * t;
	REAL z = (fColorMap[low][2]) * (1 - t) + (fColorMap[high][2]) * t;
	REAL3 color = make_REAL3(x, y, z);
	return color;
}

